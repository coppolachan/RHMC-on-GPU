#include "cuda_err.cuh"
#include <stdio.h>


void cudaSafe(const char* position, hipError_t error, const char* message)
  {
  if(error!=hipSuccess) 
    { 
    fprintf(stderr,"\033[31mERROR: %s : %s : %s\033[0m\n", position, 
	    message, hipGetErrorString(error)); 
    exit(-1); 
    }
  }

void cudaCheckError(const char *position, const char *message)
     {
     hipError_t error = hipGetLastError();
     if(error!=hipSuccess) 
       {
       fprintf(stderr,"\033[31mERROR: %s : %s : %s\033[0m\n", position, 
	       message, hipGetErrorString(error)); 
       exit(-1);
       }
     } 


