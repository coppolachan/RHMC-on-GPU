#include "hip/hip_runtime.h"
#include "include/global_const.h"
#define CUDA_LOCAL
#include "cuda_rhmc.cuh"

extern "C" void cuda_init0(void)
  {
  #ifdef DEBUG_MODE
  printf("DEBUG: inside cuda_init0 ...\n");
  #endif

  size_t gauge_field_size_f = 3*sizeof(float4)*no_links;    //first two lines only of each SU(3) matrix

  // allocate & initialize gauge
  // 2 since 1double~2float
  #ifdef USE_PINNED
  float *gauge_pinned;
  cudaSafe(AT,hipHostAlloc((void**)&gauge_pinned, 2*gauge_field_size_f, hipHostMallocWriteCombined), "hipHostAlloc");
  cudaSafe(AT,hipMemcpy(gauge_pinned, gauge_field_packed, 2*gauge_field_size_f, hipMemcpyHostToHost), "hipMemcpy");
  cudaSafe(AT,hipMalloc((void**)&gauge_field_device, 2*gauge_field_size_f), "hipMalloc"); 
  cudaSafe(AT,hipMemcpy(gauge_field_device, gauge_pinned, 2*gauge_field_size_f, hipMemcpyHostToDevice), "hipMemcpy");
  cudaSafe(AT,hipHostFree(gauge_pinned), "hipHostFree");
  #else
  cudaSafe(AT,hipMalloc((void**)&gauge_field_device, 2*gauge_field_size_f), "hipMalloc");
  cudaSafe(AT,hipMemcpy(gauge_field_device, gauge_field_packed, 2*gauge_field_size_f, hipMemcpyHostToDevice), 
                 "hipMemcpy");
  #endif

  // allocate & initialize device_table
  cudaSafe(AT,hipMalloc((void**)&device_table, sizeof(int)*size*8), "hipMalloc");
  cudaSafe(AT,hipMemcpy(device_table, shift_table, sizeof(int)*size*8, hipMemcpyHostToDevice), "hipMemcpy"); 

  // allocate & initialize device_phases
  cudaSafe(AT,hipMalloc((void**)&device_phases, sizeof(int)*size*4), "hipMalloc");
  cudaSafe(AT,hipMemcpy(device_phases, eta, sizeof(int)*size*4, hipMemcpyHostToDevice), "hipMemcpy"); 


  // initialize constants
  float mass_l=(float) GlobalParams::Instance().getMass();
  double mass = GlobalParams::Instance().getMass();
  cudaSafe(AT,hipMemcpyToSymbol(HIP_SYMBOL(mass_dev), &mass_l, sizeof(float), 0, hipMemcpyHostToDevice), "hipMemcpyToSymbol");
  cudaSafe(AT,hipMemcpyToSymbol(HIP_SYMBOL(mass_d_dev), &mass, sizeof(double), 0, hipMemcpyHostToDevice), "hipMemcpyToSymbol");
  int size_l=(int) size;
  cudaSafe(AT,hipMemcpyToSymbol(HIP_SYMBOL(size_dev), &size_l, sizeof(int), 0, hipMemcpyHostToDevice), "hipMemcpyToSymbol");
  size_l=(int) sizeh;
  cudaSafe(AT,hipMemcpyToSymbol(HIP_SYMBOL(size_dev_h), &size_l, sizeof(int), 0, hipMemcpyHostToDevice), "hipMemcpyToSymbol");

  if (GlobalChemPotPar::Instance().UseChem()) {
    double eim_cos = GlobalChemPotPar::Instance().getEim_cos();
    double eim_sin = GlobalChemPotPar::Instance().getEim_sin();
    mass_l=(float)eim_cos;
    cudaSafe(AT,hipMemcpyToSymbol(HIP_SYMBOL(dev_eim_cos_f), &mass_l, sizeof(float), 0, hipMemcpyHostToDevice), "hipMemcpyToSymbol");
    mass_l=(float)eim_sin;
    cudaSafe(AT,hipMemcpyToSymbol(HIP_SYMBOL(dev_eim_sin_f), &mass_l, sizeof(float), 0, hipMemcpyHostToDevice), "hipMemcpyToSymbol");
    cudaSafe(AT,hipMemcpyToSymbol(HIP_SYMBOL(dev_eim_cos_d), &eim_cos, sizeof(double), 0, hipMemcpyHostToDevice), "hipMemcpyToSymbol");
    cudaSafe(AT,hipMemcpyToSymbol(HIP_SYMBOL(dev_eim_sin_d), &eim_sin, sizeof(double), 0, hipMemcpyHostToDevice), "hipMemcpyToSymbol");
  }

  #ifdef DEBUG_MODE
  printf("\tterminated cuda_init0\n");
  #endif
  }


extern "C" void cuda_init1(void)
  {
  #ifdef DEBUG_MODE
  printf("DEBUG: inside cuda_init1 ...\n");
  #endif

  size_t vector_size_f   = sizeof(float2)*3*size;           // 2(complex)*3(su3_vector)

  // allocate & initialize mf_device
  // again 2 since 1double~2float
  cudaSafe(AT,hipMalloc((void**)&mf_device, 2*no_ps*vector_size_f), "hipMalloc"); 
  cudaSafe(AT,hipMemset(mf_device, 0, 2*no_ps*vector_size_f), "hipMemset");  // initialize even and odd to 0
  for(int ps=0; ps<no_ps; ps++)                  // copy the even entries from host
     {
     // 1st float
     cudaSafe(AT,hipMemcpy(mf_device          + ps*3*size, chi_packed          + ps*3*size, 
                                    size*sizeof(float), hipMemcpyHostToDevice), "hipMemcpy");
     cudaSafe(AT,hipMemcpy(mf_device +   size + ps*3*size, chi_packed +   size + ps*3*size, 
                                    size*sizeof(float), hipMemcpyHostToDevice), "hipMemcpy");
     cudaSafe(AT,hipMemcpy(mf_device + 2*size + ps*3*size, chi_packed + 2*size + ps*3*size, 
                                    size*sizeof(float), hipMemcpyHostToDevice), "hipMemcpy");

     // 2nd float
     cudaSafe(AT,hipMemcpy(mf_device          + ps*3*size + no_ps*3*size, chi_packed          + ps*3*size + no_ps*3*size, 
                                    size*sizeof(float), hipMemcpyHostToDevice), "hipMemcpy");
     cudaSafe(AT,hipMemcpy(mf_device +   size + ps*3*size + no_ps*3*size, chi_packed +   size + ps*3*size + no_ps*3*size, 
                                    size*sizeof(float), hipMemcpyHostToDevice), "hipMemcpy");
     cudaSafe(AT,hipMemcpy(mf_device + 2*size + ps*3*size + no_ps*3*size, chi_packed + 2*size + ps*3*size + no_ps*3*size, 
                                    size*sizeof(float), hipMemcpyHostToDevice), "hipMemcpy");

     }

  // allocate & initialize to zero smf_device (even & odd)
  // again 2 since 1double~2float
  cudaSafe(AT,hipMalloc((void**)&smf_device, 2*no_ps*max_approx_order*vector_size_f), "hipMalloc"); 
  cudaSafe(AT,hipMemset(smf_device, 0, 2*no_ps*max_approx_order*vector_size_f), "hipMemset"); 

  // allocate & initialize ipdot_device
  cudaSafe(AT,hipMalloc((void**)&ipdot_device, 8*no_links*sizeof(float)), "hipMalloc"); 
  cudaSafe(AT,hipMemcpy(ipdot_device, ipdot_packed, 8*no_links*sizeof(float), hipMemcpyHostToDevice), "hipMemcpy"); 

  // allocate & initialize momenta_device
  cudaSafe(AT,hipMalloc((void**)&momenta_device, 8*no_links*sizeof(float)), "hipMalloc"); 
  cudaSafe(AT,hipMemcpy(momenta_device, momenta_packed, 8*no_links*sizeof(float), hipMemcpyHostToDevice), "hipMemcpy");

  #ifdef DEBUG_MODE
  printf("\tterminated cuda_init1\n");
  #endif
  }


extern "C" void cuda_meas_init(void)
  {
  #ifdef DEBUG_MODE
  printf("DEBUG: inside cuda_meas_init ...\n");
  #endif

  size_t vector_size_f   = sizeof(float2)*3*size;           // 2(complex)*3(su3_vector)

  // allocate & initialize to zero mf_device
  // again 2 since 1double~2float
  cudaSafe(AT,hipMalloc((void**)&mf_device, 2*no_ps*vector_size_f), "hipMalloc"); 
  cudaSafe(AT,hipMemset(mf_device, 0, 2*no_ps*vector_size_f), "hipMemset");  // initialize even and odd to 0

  // allocate & initialize to zero smf_device (even & odd)
  // again 2 since 1double~2float
  cudaSafe(AT,hipMalloc((void**)&smf_device, 2*no_ps*max_approx_order*vector_size_f), "hipMalloc"); 
  cudaSafe(AT,hipMemset(smf_device, 0, 2*no_ps*max_approx_order*vector_size_f), "hipMemset"); 

  // 1st float
  cudaSafe(AT,hipMemcpy(mf_device         , simple_fermion_packed         , 
                                 size*sizeof(float), hipMemcpyHostToDevice), "hipMemcpy");
  cudaSafe(AT,hipMemcpy(mf_device +   size, simple_fermion_packed +   size, 
                                 size*sizeof(float), hipMemcpyHostToDevice), "hipMemcpy");
  cudaSafe(AT,hipMemcpy(mf_device + 2*size, simple_fermion_packed + 2*size, 
                                 size*sizeof(float), hipMemcpyHostToDevice), "hipMemcpy");

  // 2nd float
  cudaSafe(AT,hipMemcpy(mf_device         + no_ps*3*size, simple_fermion_packed           + 3*size, 
                                 size*sizeof(float), hipMemcpyHostToDevice), "hipMemcpy");
  cudaSafe(AT,hipMemcpy(mf_device +  size + no_ps*3*size, simple_fermion_packed +    size + 3*size, 
                                 size*sizeof(float), hipMemcpyHostToDevice), "hipMemcpy");
  cudaSafe(AT,hipMemcpy(mf_device + 2*size + no_ps*3*size, simple_fermion_packed + 2*size + 3*size, 
                                 size*sizeof(float), hipMemcpyHostToDevice), "hipMemcpy");


  #ifdef DEBUG_MODE
  printf("\tterminated cuda_meas_init\n");
  #endif
  }


extern "C" void cuda_meas_end(void)
  {
  #ifdef DEBUG_MODE
  printf("DEBUG: inside cuda_meas_end ...\n");
  #endif

  const int offset_2f=3*size*1*no_ps;  // num_shifts=1

  // 1st float
  cudaSafe(AT,hipMemcpy(simple_fermion_packed,          smf_device, 
                                     sizeh*sizeof(float2), hipMemcpyDeviceToHost), "hipMemcpy");
  cudaSafe(AT,hipMemcpy(simple_fermion_packed +   size, smf_device +   size, 
                                     sizeh*sizeof(float2), hipMemcpyDeviceToHost), "hipMemcpy");
  cudaSafe(AT,hipMemcpy(simple_fermion_packed + 2*size, smf_device + 2*size, 
                                     sizeh*sizeof(float2), hipMemcpyDeviceToHost), "hipMemcpy");
  // 2nd float
  cudaSafe(AT,hipMemcpy(simple_fermion_packed +          3*size, smf_device +          offset_2f, 
                                     sizeh*sizeof(float2), hipMemcpyDeviceToHost), "hipMemcpy");
  cudaSafe(AT,hipMemcpy(simple_fermion_packed +   size + 3*size, smf_device +   size + offset_2f, 
                                     sizeh*sizeof(float2), hipMemcpyDeviceToHost), "hipMemcpy");
  cudaSafe(AT,hipMemcpy(simple_fermion_packed + 2*size + 3*size, smf_device + 2*size + offset_2f, 
                                     sizeh*sizeof(float2), hipMemcpyDeviceToHost), "hipMemcpy");

  cudaSafe(AT,hipFree(mf_device), "hipFree");
  cudaSafe(AT,hipFree(smf_device), "hipFree");

 
  #ifdef DEBUG_MODE
  printf("\tterminated cuda_meas_end\n");
  #endif
  }



extern "C" void cuda_end(void)
  {
  #ifdef DEBUG_MODE
  printf("DEBUG: inside cuda_end ...\n");
  #endif

  cudaSafe(AT,hipMemcpy(gauge_field_packed, gauge_field_device, 2*3*no_links*sizeof(float4), hipMemcpyDeviceToHost), 
                                                                                                            "hipMemcpy");
  cudaSafe(AT,hipMemcpy(momenta_packed, momenta_device, 8*no_links*sizeof(float), hipMemcpyDeviceToHost), "hipMemcpy");
  cudaSafe(AT,hipMemcpy(ipdot_packed, ipdot_device, sizeof(float)*no_links*8, hipMemcpyDeviceToHost), "hipMemcpy");

  cudaSafe(AT,hipFree(gauge_field_device), "hipFree");
  cudaSafe(AT,hipFree(device_table), "hipFree");
  cudaSafe(AT,hipFree(device_phases), "hipFree");

  cudaSafe(AT,hipFree(mf_device), "hipFree");
  cudaSafe(AT,hipFree(smf_device), "hipFree");

  cudaSafe(AT,hipFree(ipdot_device), "hipFree");
  cudaSafe(AT,hipFree(momenta_device), "hipFree");

  #ifdef DEBUG_MODE
  printf("\tterminated cuda_end\n");
  #endif
  }


extern "C" void cuda_get_conf(void)
 {
 #ifdef DEBUG_MODE
 printf("DEBUG: inside cuda_get_conf ...\n");
 #endif

 cudaSafe(AT,hipMemcpy(gauge_field_packed, gauge_field_device, 2*12*no_links*sizeof(float), hipMemcpyDeviceToHost), 
       "hipMemcpy");

 #ifdef DEBUG_MODE
 printf("\tterminated cuda_get_conf ...\n");
 #endif
 }


extern "C" void cuda_get_momenta(void)
 {
 #ifdef DEBUG_MODE
 printf("DEBUG: inside cuda_get_momenta ...\n");
 #endif

 cudaSafe(AT,hipMemcpy(momenta_packed, momenta_device, 8*no_links*sizeof(float), hipMemcpyDeviceToHost), "hipMemcpy");

 #ifdef DEBUG_MODE
 printf("\tterminated cuda_get_momenta ...\n");
 #endif
 }


extern "C" void cuda_put_momenta(void)
 {
 #ifdef DEBUG_MODE
 printf("DEBUG: inside cuda_put_momenta ...\n");
 #endif

 cudaSafe(AT,hipMemcpy(momenta_device, momenta_packed, 8*no_links*sizeof(float), hipMemcpyHostToDevice), "hipMemcpy");

 #ifdef DEBUG_MODE
 printf("\tterminated cuda_put_momenta ...\n");
 #endif
 }


// Include all kernels

#include "cuda_tool_kernels.h"

#include "cuda_dslash_kernels.h"

#include "cuda_inversion_kernels.h"

#include "cuda_update_kernels.h"


